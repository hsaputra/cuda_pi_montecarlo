
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

using namespace std;

// Cont
__global__ void count_samples_in_circles_slow(float* d_randNumsX, float* d_randNumsY, int* d_countInBlocks, int nsamples)
{

  __shared__ int shared_blocks_slow[500];

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterates through
  int inCircle = 0;
  for (int i = index*1000; i < 1000 * (index + 1) ; i++) {
    float xValue = d_randNumsX[i];
    float yValue = d_randNumsY[i];

    if (xValue*xValue + yValue*yValue < 1.0f) {
      inCircle++;
    }
  }

  shared_blocks_slow[threadIdx.x] = inCircle;

  __syncthreads();

  if (threadIdx.x == 0) {
    int totalInCircleForABlock = 0;
    for (int j = 0; j < blockDim.x; j++) {
      totalInCircleForABlock += shared_blocks_slow[j];
    }
    d_countInBlocks[blockIdx.x] = totalInCircleForABlock;
  }

}

// Coales
__global__ void count_samples_in_circles(float* d_randNumsX, float* d_randNumsY, int* d_countInBlocks, int nsamples)
{

  __shared__ int shared_blocks[500];

  int index = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterates through 
  int inCircle = 0;
  int stride = gridDim.x * blockDim.x;
  for (int i = index; i < nsamples; i+=stride) {
    float xValue = d_randNumsX[i];
    float yValue = d_randNumsY[i];

    if (xValue*xValue + yValue*yValue < 1.0f) {
      inCircle++;
    }
  }

  shared_blocks[threadIdx.x] = inCircle;

  __syncthreads();

  if (threadIdx.x == 0) {
    int totalInCircleForABlock = 0;
    for (int j = 0; j < blockDim.x; j++) {
      totalInCircleForABlock += shared_blocks[j];
    }
    d_countInBlocks[blockIdx.x] = totalInCircleForABlock;
  }

}

int nsamples = 1e8;

int main(void)
{
    // allocate space to hold random values
    vector<float> h_randNumsX(nsamples);
    vector<float> h_randNumsY(nsamples);

    srand(time(NULL)); // seed with system clock

    //Initialize vector with random values
    for (int i = 0; i < h_randNumsX.size(); ++i) {
        h_randNumsX[i] = float(rand()) / RAND_MAX;
        h_randNumsY[i] = float(rand()) / RAND_MAX;
    }

    // Send random values to the GPU
    size_t size = nsamples * sizeof(float);
    float* d_randNumsX;
    float* d_randNumsY;
    hipMalloc(&d_randNumsX, size);
    hipMalloc(&d_randNumsY, size);

    hipMemcpy(d_randNumsX, &h_randNumsX.front(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_randNumsY, &h_randNumsY.front(), size, hipMemcpyHostToDevice);


    int threadsPerBlock = 500;
    int num_blocks = nsamples / (1000 * threadsPerBlock);
    int* d_countInBlocks;
    size_t countBlocks = num_blocks * sizeof(int);
    hipMalloc(&d_countInBlocks, countBlocks);

    hipEvent_t start, stop;

    // START CONT
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // CALL KERNEL  
    count_samples_in_circles_slow<<<num_blocks, threadsPerBlock>>>(d_randNumsX, d_randNumsY, d_countInBlocks, nsamples);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    float elapsedTime_0;
    hipEventElapsedTime(&elapsedTime_0, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemset(d_countInBlocks, 0, sizeof(int));

    cout << "Elapsed time Slow: " << elapsedTime_0 << endl;


    // START COALES
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    // CALL KERNEL  
    count_samples_in_circles<<<num_blocks, threadsPerBlock>>>(d_randNumsX, d_randNumsY, d_countInBlocks, nsamples);

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    cout << "Elapsed time Coales: " << elapsedTime << endl;

    // Return back the vector from device to host
    int* h_countInBlocks = new int[num_blocks];
    hipMemcpy(h_countInBlocks, d_countInBlocks, countBlocks, hipMemcpyDeviceToHost);

    int nsamples_in_circle = 0;
    for (int i = 0 ; i < num_blocks; i++) {
      //cout << "Value in block " + i << " is " << h_countInBlocks[i] << endl;
      nsamples_in_circle = nsamples_in_circle + h_countInBlocks[i];
    }
    
    hipFree(d_randNumsX);
    hipFree(d_randNumsY);
    hipFree(d_countInBlocks);


    // fraction that fell within (quarter) of unit circle
    float estimatedValue = 4.0 * float(nsamples_in_circle) / nsamples;

    cout << "Estimated Value: " << estimatedValue << endl;
}

