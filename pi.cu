
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

using namespace std;

// Create a kernel to estimate pi
__global__ 
void count_samples_in_circles(float* d_randNumsX, float* d_randNumsY, int* d_countInBlocks, int num_blocks, int nsamples) 
{

  __shared__ int shared_blocks[500];

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * num_blocks;

  // Iterates through
  int inCircle = 0;  
  for (int i = index; i < nsamples; i+= stride) {    
    float xValue = d_randNumsX[i];    
    float yValue = d_randNumsY[i];

    if (xValue*xValue + yValue*yValue <= 1.0f) {
      inCircle++;    
    }  
  }
  
  shared_blocks[threadIdx.x] = inCircle;

  __syncthreads();

  // Pick thread 0 for each block to collect all points from each Thread.
  if (threadIdx.x == 0) 
  {    
    int totalInCircleForABlock = 0;    
    for (int j = 0; j < blockDim.x; j++) 
    {      
      totalInCircleForABlock += shared_blocks[j];    
    }
    d_countInBlocks[blockIdx.x] = totalInCircleForABlock;  
  }
}

int nsamples = 1e8;

int main(void) {   
  // allocate space to hold random values    
  vector<float> h_randNumsX(nsamples);    
  vector<float> h_randNumsY(nsamples);
  srand(time(NULL)); // seed with system clock
    
  //Initialize vector with random values    
  for (int i = 0; i < h_randNumsX.size(); ++i) 
  {        
    h_randNumsX[i] = float(rand()) / RAND_MAX;        
    h_randNumsY[i] = float(rand()) / RAND_MAX;    
  }
  
  // Send random values to the GPU    
  size_t size = nsamples * sizeof(float);    
  float* d_randNumsX;    
  float* d_randNumsY;    
  
  hipMalloc(&d_randNumsX, size);  
  hipMalloc(&d_randNumsY, size);
  hipMemcpy(d_randNumsX, &h_randNumsX.front(), size, hipMemcpyHostToDevice);    
  hipMemcpy(d_randNumsY, &h_randNumsY.front(), size, hipMemcpyHostToDevice);
  
  // Launch kernel to count samples that fell inside unit circle    
  int threadsPerBlock = 500;
  int num_blocks = nsamples / (1000 * threadsPerBlock);
  size_t countBlocks = num_blocks * sizeof(int);
 
  int* d_countInBlocks;
  hipMalloc(&d_countInBlocks, countBlocks);

  // CALL KERNEL  
  count_samples_in_circles<<<num_blocks, threadsPerBlock>>>(d_randNumsX, d_randNumsY, d_countInBlocks, num_blocks, nsamples);
  if ( hipSuccess != hipGetLastError() )
    cout << "Error!\n";

  hipDeviceSynchronize();

  // Return back the vector from device to host
  int* h_countInBlocks = new int[num_blocks];
  hipMemcpy(h_countInBlocks, d_countInBlocks, countBlocks, hipMemcpyDeviceToHost);

  int nsamples_in_circle = 0;
  for (int i = 0 ; i < num_blocks; i++) {
    //cout << "Value in block " + i << " is " << h_countInBlocks[i] << endl;
    nsamples_in_circle = nsamples_in_circle + h_countInBlocks[i];
  }

  hipFree(d_randNumsX);
  hipFree(d_randNumsY);
  hipFree(d_countInBlocks);

  // fraction that fell within (quarter) of unit circle
  float estimatedValue = 4.0 * float(nsamples_in_circle) / nsamples;

  cout << "Estimated Value: " << estimatedValue << endl;
}

